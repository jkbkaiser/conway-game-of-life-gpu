#include "hip/hip_runtime.h"
/*
Conway's Game of Life rules:

1. Survival:
   - A live cell with 2 or 3 live neighbors stays alive.

2. Birth:
   - A dead cell with exactly 3 live neighbors becomes alive.

3. Death:
   - A live cell with fewer than 2 live neighbors dies (underpopulation).
   - A live cell with more than 3 live neighbors dies (overpopulation).
*/

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#include <iostream>
#include <math.h>
#include <filesystem>
#include <string.h>
#include <random>
#include <chrono>

const std::filesystem::path IMG_DIR = "./images";
 
int write_grid_to_img(int *grid, int rows, int cols, int cell_size, int iter)
{
  int n_rows = rows * cell_size;
  int n_cols = cols * cell_size;

  uint8_t* pixels = new uint8_t[n_rows * n_cols * 3];

  for (int row = 0; row < rows; ++row) {
    for (int col = 0; col < cols; ++col) {
      int alive = grid[row * cols + col];
      uint8_t color = alive ? 255 : 0;

      for (int dy = 0; dy < cell_size; ++dy) {
        for(int dx = 0; dx < cell_size; ++dx) {
          int y = row * cell_size + dy;
          int x = col * cell_size + dx;
          int pixel = (y * n_cols + x) * 3;

          pixels[pixel + 0] = color;
          pixels[pixel + 1] = color;
          pixels[pixel + 2] = color;
        }
      }
    }
  }

  std::filesystem::path filename = IMG_DIR / ("frame_" + std::to_string(iter) + ".png");
  stbi_write_png(filename.c_str(), n_cols, n_rows, 3, pixels, n_cols * 3);
  return 0;
}

__global__
void compute_step_cuda(int *gridA, int *gridB, int rows, int cols)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = index; i < rows * cols; i+=stride) {
    int row = i / cols;
    int col = i % cols;

    int neighbors = 0;

    for (int dy = -1; dy <= 1; ++dy) {
      for (int dx = -1; dx <= 1; ++dx) {
        if (dx == 0 && dy == 0) continue;

        int x = col + dx;
        int y = row + dy;

        if (x < 0 || x >= cols || y < 0 || y >= rows) continue;

        int alive = gridA[y * cols + x];

        if (alive == 1) {
          ++neighbors;
        }
      }
    }

    int gridIdx = row * cols + col;
    int alive = gridA[gridIdx];

    if (alive) {
      if (neighbors < 2 || 3 < neighbors) {
        gridB[gridIdx] = 0;
      } else {
        gridB[gridIdx] = 1;
      }
    } else  {
      if (neighbors == 3) {
        gridB[gridIdx] = 1;
      } else {
        gridB[gridIdx] = 0;
      }
    }
  }
}

void compute_step_cpu(int *gridA, int *gridB, int rows, int cols)
{
  for (int row = 0; row < rows; ++row) {
    for (int col = 0; col < cols; ++col) {
      int neighbors = 0;

      for (int dy = -1; dy <= 1; ++dy) {
        for (int dx = -1; dx <= 1; ++dx) {
          if (dx == 0 && dy == 0) continue;

          int x = col + dx;
          int y = row + dy;

          if (x < 0 || x >= cols || y < 0 || y >= rows) continue;

          int alive = gridA[y * cols + x];

          if (alive == 1) {
            ++neighbors;
          }
        }
      }

      int index = row * cols + col;
      int alive = gridA[index];

      if (alive) {
        if (neighbors < 2 || 3 < neighbors) {
          gridB[index] = 0;
        } else {
          gridB[index] = 1;
        }
      } else  {
        if (neighbors == 3) {
          gridB[index] = 1;
        } else {
          gridB[index] = 0;
        }
      }
    }
  }
}

int main(void)
{
  std::filesystem::create_directory(IMG_DIR);

  bool useGPU = true;
  bool saveImgs = true;
  int scale = 1;

  // 1024 vals
  float aspectRatio = 1.0f;
  float imgSize = 1024;

  int cols = imgSize;
  int rows = imgSize / aspectRatio;

  int iters = 200;

  int *gridA = new int[rows * cols];
  int *gridB = new int[rows * cols];

  std::random_device rd;
  std::mt19937 gen(rd());
  std::bernoulli_distribution dist(0.2);

  for (int i = 0; i < rows * cols; ++i) {
    gridA[i] = dist(gen);
  }

  if (saveImgs) {
    write_grid_to_img(gridA, rows, cols, scale, 0);
  }

  int blockSize = 256;
  int gridSize = (rows * cols + blockSize - 1) / blockSize;

  int *gridADevice;
  int *gridBDevice;
  int gridNumBytes = rows * cols * sizeof(int);

  if (useGPU) {
    hipMalloc(&gridADevice, gridNumBytes);
    hipMalloc(&gridBDevice, gridNumBytes);
    hipMemcpy(gridADevice, gridA, gridNumBytes, hipMemcpyHostToDevice);
  }

  float avgTime = 0.0f;

  for (int i = 1; i < iters; ++i) {
    float progress = float(i) / iters;
    int barWidth = 50;

    std::cout << "\r" << i << "/" << iters << " [";
    int pos = barWidth * progress;
    for (int j = 0; j < barWidth; ++j) {
        std::cout << (j < pos ? "=" : (j == pos ? ">" : " "));
    }
    std::cout << "] " << int(progress * 100.0) << "%" << std::flush;

    auto start = std::chrono::high_resolution_clock::now();

    // CPU
    if (!useGPU) {
      compute_step_cpu(gridA, gridB, rows, cols);
    }

    // GPU
    if (useGPU) {
      compute_step_cuda<<<gridSize, blockSize>>>(gridADevice, gridBDevice, rows, cols);
      hipDeviceSynchronize();
      hipMemcpy(gridB, gridBDevice, gridNumBytes, hipMemcpyDeviceToHost);
    }

    // Measure time
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> elapsed = end - start;
    avgTime += elapsed.count();

    if (saveImgs) {
      write_grid_to_img(gridB, rows, cols, scale, i);
    }

    if (useGPU) {
      std::swap(gridADevice, gridBDevice);
    } else {
      std::swap(gridA, gridB);
    }
  }

  std::cout << "\nAvg compute time: " << avgTime / iters << " ms" << std::endl;

  if (useGPU) {
    hipFree(gridADevice);
    hipFree(gridBDevice);
  }

  delete[] gridA;
  delete[] gridB;

  return 0;
}
